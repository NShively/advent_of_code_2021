#include "hip/hip_runtime.h"
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>


/**
 * CUDA Kernel Device Code
 * Just checks the value in the array before it and increments a counter if it 
 */
__global__ void 
vectorGreaterThan(const int *A, bool* B, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i <= 0 || i >= size) {
        return;
    }
    if (A[i] > A[i-1]) {
        B[i-1] = true;
    }
    
}

__global__ void
vector3dSum(const int *A, int* B, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i <= 1 || i >= size) {
        return;
    }
    B[i-2] = A[i] + A[i-1] + A[i-2];
//    printf("test val b: i[%d] B[%d]\n", i, B[i-2]); 
}

int main() {
    std::vector<int> inData;
    std::ifstream File;
    File.open("input.txt");
    while(!File.eof())
    {
        int p = 0;
        File >> p;
        if (File.eof()) break;
        inData.push_back(p);
    }
    int numElements = inData.size();
    size_t sizeInput = numElements * sizeof(int);
    size_t sizeOutput = (numElements-1) * sizeof(bool);
    size_t sizeOutput2 = 2*(numElements-2) * sizeof(int);
    int *host_input = (int*)malloc(sizeInput);
    bool *host_output = (bool*)malloc(sizeOutput);
    host_input = inData.data();   
    hipError_t err = hipSuccess;
    int * d_input = NULL;
    err = hipMalloc((void**) &d_input, sizeInput);
    if (err != hipSuccess) {
        std::cout<< "Errormallocing device input!" << std::endl;
        std::cout << hipGetErrorString(err) << std::endl;
    }

    bool * d_output = NULL;
    err = hipMalloc((void**) &d_output, sizeOutput);
    if (err != hipSuccess) {
        std::cout<< "Errormallocing device input!" << std::endl;
        std::cout << hipGetErrorString(err) << std::endl;
    }

    err = hipMemcpy(d_input, host_input, sizeInput, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        std::cout << "Failed to copy from host to device: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock=256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    vectorGreaterThan<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, sizeInput);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        std::cout << "vectorGreaterThan kernel error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(host_output, d_output, sizeOutput, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cout << "Failed to copy result back to host: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    int total = 0;
    for (int i = 0; i < sizeOutput; i++) {
        if (host_output[i]) total++;
    }
    std::cout << "part A total: " << total << std::endl;

    int* host_output_pt2 = (int*)malloc(sizeOutput2);
    int* d_output_2 = NULL;
    err = hipMalloc((void**)&d_output_2, sizeOutput2);
    if (err != hipSuccess)
    {
        std::cout << "error pt2 malloc " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    vector3dSum<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output_2, numElements);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cout << "error in pt2 kernl: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(host_output_pt2, d_output_2, sizeOutput2, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cout << "error copy back to host pt2: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    int total2 = 0;
    for (int i = 1; i < numElements-2; i++)
    {
        if (host_output_pt2[i] > host_output_pt2[i-1]) total2++;
    }
    std::cout << "part B total: " << total2 << std::endl;
}
